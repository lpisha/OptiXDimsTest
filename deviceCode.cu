#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2022 Louis Pisha                                               //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <owl/owl.h>
#include <optix_device.h>

OPTIX_RAYGEN_PROGRAM(dimsTestRaygenProgram)(){
    uint3 lbounds = optixGetLaunchDimensions();
    uint3 lidx = optixGetLaunchIndex();
    if(lidx.x == 0 && lidx.y == 0 && lidx.z == 0){
        printf("Raygen with size (%d,%d,%d) successful\n", lbounds.x, lbounds.y, lbounds.z);
    }
}
