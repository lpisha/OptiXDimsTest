// ======================================================================== //
// Copyright 2022 Louis Pisha                                               //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <iostream>
#include "owl/owl.h"

struct DimsTestRaygenData {
    uint32_t dummy;
};

int32_t gpu_number = 0;

extern "C" char deviceCode_ptx[];

int main(int argc, char **argv){
    if(argc != 4){
        std::cout << "Usage: ./OptiXDimsTest X Y Z\nwhere X, Y, Z are positive integers representing the OptiX launch dimensions\n";
        return -1;
    }
    int x, y, z;
    char *endx, *endy, *endz;
    x = strtol(argv[1], &endx, 0);
    y = strtol(argv[2], &endy, 0);
    z = strtol(argv[3], &endz, 0);
    if(x <= 0 || y <= 0 || z <= 0 || *endx != 0 || *endy != 0 || *endz != 0){
        std::cout << "Invalid arguments\n";
        return -1;
    }
    
    OWLContext owlContext = owlContextCreate(&gpu_number, 1);
    OWLModule owlModule = owlModuleCreate(owlContext, deviceCode_ptx);
    OWLVarDecl dimsTestRaygenDataTypeDecl[] = {
        {"dummy", OWL_UINT, OWL_OFFSETOF(DimsTestRaygenData, dummy)},
        {}
    };
    OWLRayGen dimsTestRaygen = owlRayGenCreate(owlContext, owlModule, "dimsTestRaygenProgram",
        sizeof(DimsTestRaygenData), dimsTestRaygenDataTypeDecl, -1);
    owlBuildPrograms(owlContext);
    owlBuildPipeline(owlContext);
    owlRayGenSet1ui(dimsTestRaygen, "dummy", 12345);
    owlBuildSBT(owlContext);
    owlRayGenLaunch3D(dimsTestRaygen, x, y, z);
    hipError_t err = hipDeviceSynchronize();
    if(err != hipSuccess){
        std::cout << "Sync failed with error " << (int)err << "\n";
        return 1;
    }
    return 0;
}
